#include "hip/hip_runtime.h"
#include <iostream>
#include <fcl/fcl.h>
#include <limits>
#include <vector>
#include "Utils.h"

void createAlphaBotConfigurations(std::vector<Configuration> &confs, int num_confs){
  // these are the max and min values for vertices in alpha1.0/robot.obj
    float x_min = 3.72119;
    float y_min = -11.0518;
    float z_min = -0.608012;
    float x_max = 65.9453;
    float y_max = 26.0984;
    float z_max = 18.6984;

    float x_range = x_max - x_min;
    float y_range = y_max - y_min;
    float z_range = z_max - z_min;

    generateConfs(confs, -x_range/20, x_range/20,
                         -y_range/20, y_range/20,
                         -z_range/20, z_range/20,
                         num_confs);

}

//TODO: move robot to constant memory
//TODO: refactor code to minimize loads by interleaving file reads and device memory operations
int main()
{
    int device_count;
    if (hipGetDeviceCount(&device_count) != 0) std::cout << "CUDA not loaded properly" << std::endl;
    // load configurations, should have 6990 valids and 3010 invalids
    std::vector<Configuration> confs;
    readConfigurationFromFile("10,000samples.conf", confs);
    // createAlphaBotConfigurations(confs, 10000);

    //Load Robot
    std::vector<Vector3f> rob_vertices;
    std::vector<Triangle> rob_triangles;
    loadOBJFile("models/alpha1.0/robot.obj", rob_vertices, rob_triangles);
    std::cout << "robot has " << rob_vertices.size() << " vertices " <<std::endl;

    // Load Obstacle
    std::vector<Vector3f> obs_vertices;
    std::vector<Triangle> obs_triangles;
    loadOBJFile("models/alpha1.0/obstacle.obj", obs_vertices, obs_triangles);
    std::cout << "obstacle has " << obs_vertices.size() << " vertices " <<std::endl;

    Vector3f *d_rob_vertices;
    checkCudaCall(hipMalloc(&d_rob_vertices, rob_vertices.size() * sizeof(Vector3f)));
    checkCudaMem(hipMemcpy(d_rob_vertices, rob_vertices.data(), rob_vertices.size() * sizeof(Vector3f), hipMemcpyHostToDevice));
    std::cout << "have copied the robot vertices " << std::endl;

    Vector3f *d_transformed_vertices;
    checkCudaCall(hipMalloc(&d_transformed_vertices, rob_vertices.size() * sizeof(Vector3f) * confs.size()));
    std::cout << "have malloced the transformed vertices " << std::endl;

    Triangle *d_rob_triangles;
    checkCudaCall(hipMalloc(&d_rob_triangles, rob_triangles.size() * sizeof(Triangle)));
    checkCudaMem(hipMemcpy(d_rob_triangles, rob_triangles.data(), rob_triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice));
    std::cout << "have copied the robot triangles" << std::endl;

    Configuration *d_confs;
    checkCudaCall(hipMalloc(&d_confs, confs.size() * sizeof(Configuration)));
    checkCudaMem(hipMemcpy(d_confs, confs.data(), confs.size() * sizeof(Configuration), hipMemcpyHostToDevice));
    std::cout << "have copied the configurations " << std::endl;


    checkCudaCall(hipDeviceSynchronize());
    std::cout << "have synchronized" << std::endl;

    // bitshifting right by 5 is the same as dividing by 2^5 (which is 32) and rounding up
    // also technically faster not that it matters very much
    genTransformedCopies<<<(confs.size() + 31)>> 5, 32>>>(d_confs, d_rob_vertices, d_transformed_vertices, 
                                                    confs.size(), rob_vertices.size());
    
    std::cout << "have called kernel " << std::endl;

    Vector3f* transformed_vertices = new Vector3f[10000 * 792];
    std:: cout << "about to synchronize" << std::endl;
    checkCudaCall(hipDeviceSynchronize());
    std:: cout << "about to copy back vertices" << std::endl;
    checkCudaMem(hipMemcpy(transformed_vertices, d_transformed_vertices, rob_vertices.size()* confs.size() * sizeof(Vector3f), hipMemcpyDeviceToHost));
    checkCudaCall(hipDeviceSynchronize()); 
    checkCudaCall(hipFree(d_confs));
    checkCudaCall(hipFree(d_rob_triangles));
    checkCudaCall(hipFree(d_rob_vertices));
    checkCudaCall(hipFree(d_transformed_vertices));
    checkCudaCall(hipDeviceSynchronize());
    std::cout << " copied back memory and synchronized" << std::endl;

    //Load Robot
    std::vector<fcl::Vector3f> fcl_rob_vertices;
    std::vector<fcl::Triangle> fcl_rob_triangles;
    loadOBJFileFCL("models/alpha1.0/robot.obj", fcl_rob_vertices, fcl_rob_triangles);
    std::cout << "robot has " << fcl_rob_vertices.size() << " vertices " <<std::endl;

    std::shared_ptr<fcl::BVHModel<fcl::OBBRSS<float>>> rob_mesh(new fcl::BVHModel<fcl::OBBRSS<float>>);
    rob_mesh->beginModel(fcl_rob_triangles.size(), fcl_rob_vertices.size());
    rob_mesh->addSubModel(fcl_rob_vertices, fcl_rob_triangles);
    rob_mesh->endModel();
    std::cout << "loaded robot" <<std::endl;

    int num_correct = 0;
    int num_incorrect = 0;
    float total_error_incorrect = 0;
    for (int i = 0; i < confs.size(); i++){
    // for (int i = 0; i < 1; i++){
      fcl::Transform3f transform = configurationToTransform(confs[i]);
      fcl::Vector3f transformed_vertex;
      for (int j = 0; j < fcl_rob_vertices.size(); j++){
      // for (int j = 0; j < 10; j++){
        transformed_vertex = transform * fcl_rob_vertices[j];
        if (fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].x -transformed_vertex[0]) +
            fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].y -transformed_vertex[1]) +
            fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].z -transformed_vertex[2]) < 1e-5){
          num_correct++;
        } else {
          // std::cout << "gpu got " << transformed_vertices[i * fcl_rob_vertices.size() + j].x 
          //                         << " " << transformed_vertices[i * fcl_rob_vertices.size() + j].y
          //                         << " " << transformed_vertices[i * fcl_rob_vertices.size() + j].z <<std::endl;
          // std::cout << "cpu got " << transformed_vertex[0]  
          //                         << " " <<  transformed_vertex[1]
          //                         << " " <<  transformed_vertex[2] << std::endl;
          // std::cout << "Transform matrix:\n" << transform.matrix() << std::endl;
          num_incorrect++;
          total_error_incorrect += fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].x -transformed_vertex[0]) +
            fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].y -transformed_vertex[1]) +
            fabs(transformed_vertices[i * fcl_rob_vertices.size() + j].z -transformed_vertex[2]);
        }
      }  
    }
    std::cout << "num correct is " << num_correct << std::endl;
    std::cout << "num incorrect is " << num_incorrect << std::endl;
    std::cout << "avg incorrect error is " << total_error_incorrect / num_incorrect << std::endl;
    
}
