#include "hip/hip_runtime.h"
#include "generate-AABB.hu"

// generateAABBPrimitiveKernel - Basic, unoptimized parallel kernel to generate AABBs
//      - Single dimension block.
//      - Parallellizes over configurations i.e. each thread handles one configuration.
//      - Each thread loops over all vertices to calculate the AABB.
__global__ void generateAABBPrimitiveKernel(Vector3f* vertices, unsigned int numVertices, 
                    unsigned int numConfigs, AABB* botBounds) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < numConfigs)
    {
        unsigned int configOffset = tid * numVertices;
        botBounds[tid].x_min = vertices[configOffset].x;
        botBounds[tid].y_min = vertices[configOffset].y;
        botBounds[tid].z_min = vertices[configOffset].z;
        botBounds[tid].x_max = vertices[configOffset].x;
        botBounds[tid].y_max = vertices[configOffset].y;
        botBounds[tid].z_max = vertices[configOffset].z;
        for(int j = 0; j < numVertices; ++j)
        {
            botBounds[tid].x_min = min(botBounds[tid].x_min, vertices[configOffset + j].x);
            botBounds[tid].y_min = min(botBounds[tid].y_min, vertices[configOffset + j].y);
            botBounds[tid].z_min = min(botBounds[tid].z_min, vertices[configOffset + j].z);
            botBounds[tid].x_max = max(botBounds[tid].x_max, vertices[configOffset + j].x);
            botBounds[tid].y_max = max(botBounds[tid].y_max, vertices[configOffset + j].y);
            botBounds[tid].z_max = max(botBounds[tid].z_max, vertices[configOffset + j].z);
        }
    }
}

// generateAABBKernel - Optimized parallel kernel to generate AABBs
//      - Two dimension block - each yDim corresponds to one config, each xDim corresponds to vertices.
//      - Uses reduction along xDim to calculate AABBs for each configuration.
__global__ void generateAABBKernel(Vector3f* vertices, unsigned int numVertices, 
                    unsigned int numConfigs, AABB* botBounds) 
{    
    // __shared__ Vector3f partialMin[2 * AABB_BLOCK_SIZE_X];
    // __shared__ Vector3f partialMax[2 * AABB_BLOCK_SIZE_X];
    extern __shared__ Vector3f sharedMem[];
    Vector3f* partialMin = &sharedMem[0];
    Vector3f* partialMax = &sharedMem[2 * AABB_BLOCK_SIZE_X];
    AABB botBoundsLocal;

    unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int tx = threadIdx.x;
    unsigned int inputStartIdx = 2*blockIdx.x*blockDim.x*ty;
    
    unsigned int totalNumVertices = numVertices * numConfigs;

    if(ty < numConfigs)
    {
        if((inputStartIdx + tx) < totalNumVertices && tx < numVertices)
        {
            partialMin[tx] = vertices[inputStartIdx + tx];
            partialMax[tx] = vertices[inputStartIdx + tx];
            if((inputStartIdx + tx + blockDim.x) < totalNumVertices 
                && (tx + blockDim.x) < numVertices)
            {
                partialMin[tx + blockDim.x] = vertices[inputStartIdx + tx + blockDim.x];
                partialMax[tx + blockDim.x] = vertices[inputStartIdx + tx + blockDim.x];
            }
            else
            {
                partialMin[tx + blockDim.x] = Vector3f(0,0,0);
                partialMax[tx + blockDim.x] = Vector3f(0,0,0);
            }
        }
        else
        {
            partialMin[tx] = Vector3f(0,0,0);
            partialMin[tx + blockDim.x] = Vector3f(0,0,0);
            partialMax[tx] = Vector3f(0,0,0);
            partialMax[tx + blockDim.x] = Vector3f(0,0,0);
        }
        for(unsigned int stride = blockDim.x; stride >= 1; stride /= 2)
        {
            __syncthreads();
            if(tx < stride)
            {
                partialMin[tx].x = min(partialMin[tx].x, partialMin[tx + stride].x);
                partialMin[tx].y = min(partialMin[tx].y, partialMin[tx + stride].y);
                partialMin[tx].z = min(partialMin[tx].z, partialMin[tx + stride].z);
                partialMax[tx].x = max(partialMax[tx].x, partialMax[tx + stride].x);
                partialMax[tx].y = max(partialMax[tx].y, partialMax[tx + stride].y);
                partialMax[tx].z = max(partialMax[tx].z, partialMax[tx + stride].z);
            }
        }
        __syncthreads();

        //TODO: 
        // This algorithm reduces to an output size = blockDim.x for each config
        // Dynamically launch another kernel until only one block is launched?
        // Or reduce the output of this on the CPU. But that will require movement of data
        // from the device to host and back again for the broad phase.

        // The below code is a temporary solution which works only if one block is launched in the x dimension.

        if(tx == 0)
        {
            botBoundsLocal.x_min = partialMin[tx].x;
            botBoundsLocal.y_min = partialMin[tx].y;
            botBoundsLocal.z_min = partialMin[tx].z;
            botBoundsLocal.x_max = partialMax[tx].x;
            botBoundsLocal.y_max = partialMax[tx].y;
            botBoundsLocal.z_max = partialMax[tx].z;
            botBounds[ty] = botBoundsLocal;
        }
    }
}

// generateAABB- Generate AABBs for all configurations parallelly
void generateAABB(Vector3f* vertices, unsigned int numVertices, 
                    unsigned int numConfigs, AABB* botBounds) 
{
    int device_count;
    if (hipGetDeviceCount(&device_count) != 0) {
        printf("CUDA not loaded properly\n");
    } else {
        printf("CUDA loaded for %d device(s)\n", device_count);
    }

    Vector3f* d_vertices;
    hipMalloc(&d_vertices, numConfigs * numVertices * sizeof(Vector3f));
    hipMemcpy(d_vertices, vertices, numConfigs * numVertices * sizeof(Vector3f), hipMemcpyHostToDevice);

    AABB* d_bot_bounds;
    hipMalloc(&d_bot_bounds, numConfigs * sizeof(AABB));

    dim3 dimGrid(ceil((float)(numConfigs) / AABB_BLOCK_SIZE), 1, 1);
    dim3 dimBlock(AABB_BLOCK_SIZE, 1, 1);
    generateAABBPrimitiveKernel<<<dimGrid, dimBlock>>>(d_vertices, numVertices, numConfigs, d_bot_bounds);
    // dim3 dimGrid(ceil((float)(numVertices) / AABB_BLOCK_SIZE_X*2), ceil((float)(numConfigs) / AABB_BLOCK_SIZE_Y), 1);
    // dim3 dimBlock(AABB_BLOCK_SIZE_X, AABB_BLOCK_SIZE_Y, 1);
    // generateAABBKernel<<<dimGrid, dimBlock, 4 * AABB_BLOCK_SIZE_X>>>(d_vertices, numVertices, numConfigs, d_bot_bounds);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    printf("Status: %s: %s\n", hipGetErrorName(err), hipGetErrorString(err));

    // Copy the data back
    hipMemcpy(botBounds, d_bot_bounds, numConfigs * sizeof(AABB), hipMemcpyDeviceToHost);

    // Free the memory
    hipFree(d_bot_bounds);
    hipFree(d_vertices);
}

// generateAABBBaseline- Generate AABBs for all configurations serially
void generateAABBBaseline(Vector3f* vertices, unsigned int numVertices, 
                    unsigned int numConfigs, AABB* botBounds) 
{
    // Loop over every configuration
    for(int i = 0; i < numConfigs; ++i)
    {
        // Loop over every vertex in each configuration
        unsigned int configOffset = i * numVertices;
        botBounds[i].x_min = vertices[configOffset].x;
        botBounds[i].y_min = vertices[configOffset].y;
        botBounds[i].z_min = vertices[configOffset].z;
        botBounds[i].x_max = vertices[configOffset].x;
        botBounds[i].y_max = vertices[configOffset].y;
        botBounds[i].z_max = vertices[configOffset].z;
        for(int j = 0; j < numVertices; ++j)
        {
            botBounds[i].x_min = min(botBounds[i].x_min, vertices[configOffset + j].x);
            botBounds[i].y_min = min(botBounds[i].y_min, vertices[configOffset + j].y);
            botBounds[i].z_min = min(botBounds[i].z_min, vertices[configOffset + j].z);
            botBounds[i].x_max = max(botBounds[i].x_max, vertices[configOffset + j].x);
            botBounds[i].y_max = max(botBounds[i].y_max, vertices[configOffset + j].y);
            botBounds[i].z_max = max(botBounds[i].z_max, vertices[configOffset + j].z);
        }
    }
}