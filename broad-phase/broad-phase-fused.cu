#include "hip/hip_runtime.h"
#include "broad-phase-fused.hu"

__device__ Matrix4f createTransformationMatrix(Configuration config) {
    float x = config.x;
    float y = config.y;
    float z = config.z;
    float pitch = config.pitch;
    float yaw = config.yaw;
    float roll= config.roll;

    float cosB = cos(pitch);
    float sinB = sin(pitch);
    float cosA = cos(yaw);
    float sinA = sin(yaw);
    float cosC = cos(roll);
    float sinC = sin(roll);

    Matrix4f transform;
    transform.m[0][0] = cosA * cosB;
    transform.m[0][1] = cosA * sinB * sinC - sinA * cosC;
    transform.m[0][2] = cosA * sinB * cosC + sinA * sinC;
    transform.m[0][3] = x;
    transform.m[1][0] = sinA * cosB;
    transform.m[1][1] = sinA * sinB * sinC + cosA * cosC;
    transform.m[1][2] = sinA * sinB * cosC - cosA * sinC;
    transform.m[1][3] = y;
    transform.m[2][0] = -sinB;
    transform.m[2][1] = cosB * sinC;
    transform.m[2][2] = cosB * cosC;
    transform.m[2][3] = z;
    transform.m[3][0] = 0;
    transform.m[3][1] = 0;
    transform.m[3][2] = 0;
    transform.m[3][3] = 1;

    return transform;
}

__device__ Vector3f transformVector(Vector3f v, Matrix4f M) {
    // Create a 4D homogeneous vector from the 3D vector
    float v_h[4] = {v.x, v.y, v.z, 1};

    // Compute the transformed 4D vector by matrix multiplication
    float v_h_prime[4] = {0};
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            v_h_prime[i] += M.m[i][j] * v_h[j];
        }
    }

    // Convert the transformed 4D vector back to a 3D vector
    Vector3f v_prime = {
        v_h_prime[0] / v_h_prime[3],
        v_h_prime[1] / v_h_prime[3],
        v_h_prime[2] / v_h_prime[3]
    };

    return v_prime;
}

// Check if two objects are colliding along a certain dimension
inline __host__ __device__ bool dimensionCollides(float fstMin, float fstMax, float sndMin, float sndMax) {
    // Done without any control divergence!
    return fstMin <= sndMax && sndMin <= fstMax;
}

#define MAX_NUM_ROBOT_VERTICES 1000
__constant__ Vector3f base_robot_vertices[MAX_NUM_ROBOT_VERTICES];

__global__ void broadPhaseFusedKernel(Configuration *configs, const AABB *obstacle,
                                     bool *valid_conf, const int num_configs, const int num_robot_vertices)
{
    size_t config_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(config_idx >= num_configs) return;

    Matrix4f transform_matrix = createTransformationMatrix(configs[config_idx]);

    AABB bot_bounds_local;
    bot_bounds_local.x_min = base_robot_vertices[0].x;
    bot_bounds_local.y_min = base_robot_vertices[0].y;
    bot_bounds_local.z_min = base_robot_vertices[0].z;
    bot_bounds_local.x_max = base_robot_vertices[0].x;
    bot_bounds_local.y_max = base_robot_vertices[0].y;
    bot_bounds_local.z_max = base_robot_vertices[0].z;

    Vector3f transformed_robot_vertex;
    for(int vertex_idx = 0; vertex_idx < num_robot_vertices; ++vertex_idx)
    {      
      transformed_robot_vertex = transformVector(base_robot_vertices[vertex_idx], transform_matrix);
      
      bot_bounds_local.x_min = min(bot_bounds_local.x_min, transformed_robot_vertex.x);
      bot_bounds_local.y_min = min(bot_bounds_local.y_min, transformed_robot_vertex.y);
      bot_bounds_local.z_min = min(bot_bounds_local.z_min, transformed_robot_vertex.z);
      bot_bounds_local.x_max = max(bot_bounds_local.x_max, transformed_robot_vertex.x);
      bot_bounds_local.y_max = max(bot_bounds_local.y_max, transformed_robot_vertex.y);
      bot_bounds_local.z_max = max(bot_bounds_local.z_max, transformed_robot_vertex.z);
    } 
    // bot_bounds[config_idx] = bot_bounds_local;

    // Due to the massive reuse it's fastest to store the obstacle AABB in registers
    AABB obstacleReg = *obstacle;
    bool isNotValid = 
            dimensionCollides(obstacleReg.x_min, obstacleReg.x_max, bot_bounds_local.x_min, bot_bounds_local.x_max) &&
            dimensionCollides(obstacleReg.y_min, obstacleReg.y_max, bot_bounds_local.y_min, bot_bounds_local.y_max) &&
            dimensionCollides(obstacleReg.z_min, obstacleReg.z_max, bot_bounds_local.z_min, bot_bounds_local.z_max);
    valid_conf[config_idx] = !isNotValid;
}

void broadPhaseFused(std::vector<Configuration> &configs, bool *valid_conf)
{
    int device_count;
    if (hipGetDeviceCount(&device_count) != 0) std::cout << "CUDA not loaded properly" << std::endl;

    //Load Robot
    std::vector<Vector3f> rob_vertices;
    std::vector<Triangle> rob_triangles;
    loadOBJFile(ROB_FILE, rob_vertices, rob_triangles);
    std::cout << "Robot has " << rob_vertices.size() << " vertices " <<std::endl;

    //Load Obstacles
    std::vector<Vector3f> obs_vertices;
    std::vector<Triangle> obs_triangles;
    loadOBJFile(OBS_FILE, obs_vertices, obs_triangles);
    std::cout << "Obstacle has " << obs_vertices.size() << " vertices " <<std::endl;

    //Load robot vertices to constant memory
    checkCudaMem(hipMemcpyToSymbol(HIP_SYMBOL(base_robot_vertices), rob_vertices.data(), rob_vertices.size() * sizeof(Vector3f)));
    std::cout << "Copied the robot vertices " << std::endl;

    Configuration *d_configs;
    checkCudaCall(hipMalloc(&d_configs, configs.size() * sizeof(Configuration)));
    checkCudaMem(hipMemcpy(d_configs, configs.data(), configs.size() * sizeof(Configuration), hipMemcpyHostToDevice));
    std::cout << "Copied the configurations " << std::endl;

    // AABB* d_bot_bounds;
    // checkCudaCall(hipMalloc(&d_bot_bounds, configs.size() * sizeof(AABB)));
    // std::cout << "Malloced the AABBs " << std::endl;

    // Move obstacle to AABB (on CPU since we only have 1)
    AABB *obstacle_AABB = new AABB();
    generateAABBBaseline(obs_vertices.data(), obs_vertices.size(), 1, obstacle_AABB);

    bool *valid_conf_d;
    AABB *obstacle_AABB_d;
    checkCudaCall(hipMalloc(&valid_conf_d, configs.size() * sizeof(bool)));
    checkCudaCall(hipMalloc(&obstacle_AABB_d, sizeof(AABB)));
    checkCudaCall(hipMemcpy(obstacle_AABB_d, obstacle_AABB, sizeof(AABB), hipMemcpyHostToDevice));

    dim3 dimGridTransformKernel(ceil((float)(configs.size()) / TRANSFORM_BLOCK_SIZE), 1, 1);
    dim3 dimBlockTransformKernel(TRANSFORM_BLOCK_SIZE, 1, 1);
    broadPhaseFusedKernel<<<dimGridTransformKernel, dimBlockTransformKernel>>>(d_configs, obstacle_AABB_d, valid_conf_d,
                                                    configs.size(), rob_vertices.size());
    checkCudaCall(hipDeviceSynchronize());

    // broadPhase(configs.size(), d_bot_bounds, obstacle_AABB_d, valid_conf_d);

    std::cout << "Completed kernel execution" << std::endl;
    
    // checkCudaCall(hipDeviceSynchronize());
    std::cout << "Synchronized" << std::endl;

    std:: cout << "Copying back results" << std::endl;
    // hipMemcpy(bot_bounds, d_bot_bounds, configs.size() * sizeof(AABB), hipMemcpyDeviceToHost);
    hipMemcpy(valid_conf, valid_conf_d, configs.size() * sizeof(bool), hipMemcpyDeviceToHost);

    checkCudaCall(hipFree(d_configs));
    // checkCudaCall(hipFree(d_bot_bounds));
    checkCudaCall(hipFree(obstacle_AABB_d));
    checkCudaCall(hipFree(valid_conf_d));
    std::cout << "Copied back memory and synchronized" << std::endl;
}