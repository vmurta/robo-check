#include "hip/hip_runtime.h"
#include "../broad-phase/broad-phase-fused.hu"

// Set -DLOCAL_TESTING=1 to run CPU tests on local machine (not on rai)

#if(LOCAL_TESTING == 1)
#include <fcl/fcl.h>
#include "../Utils.h"
#endif

#if(LOCAL_TESTING == 1)
inline bool verticesEqual(const Vector3f &v1, const fcl::Vector3f &v2){
  return (fabs(v1.x -v2[0]) +
            fabs(v1.y -v2[1]) +
            fabs(v1.z -v2[2]) < 1e-5);
  // return false;
}

void generateAABBBaseline_fcl(fcl::Vector3f* vertices, unsigned int numVertices, 
                    unsigned int numConfigs, AABB* botBounds) 
{
    // Loop over every configuration
    for(int i = 0; i < numConfigs; ++i)
    {
        // Loop over every vertex in each configuration
        unsigned int configOffset = i * numVertices;
        botBounds[i].x_min = vertices[configOffset][0];
        botBounds[i].y_min = vertices[configOffset][1];
        botBounds[i].z_min = vertices[configOffset][2];
        botBounds[i].x_max = vertices[configOffset][0];
        botBounds[i].y_max = vertices[configOffset][1];
        botBounds[i].z_max = vertices[configOffset][2];
        for(int j = 0; j < numVertices; ++j)
        {
            botBounds[i].x_min = min(botBounds[i].x_min, vertices[configOffset + j][0]);
            botBounds[i].y_min = min(botBounds[i].y_min, vertices[configOffset + j][1]);
            botBounds[i].z_min = min(botBounds[i].z_min, vertices[configOffset + j][2]);
            botBounds[i].x_max = max(botBounds[i].x_max, vertices[configOffset + j][0]);
            botBounds[i].y_max = max(botBounds[i].y_max, vertices[configOffset + j][1]);
            botBounds[i].z_max = max(botBounds[i].z_max, vertices[configOffset + j][2]);
        }
    }
}
#endif

bool verify_generateAABB(AABB* botBoundsBaseline, AABB* botBoundsParallel, const int numConfigs)
{   
    int num_correct = 0;
    int num_incorrect = 0;
    float running_error = 0;
    for(int i = 0; i < numConfigs; ++i) 
    {   
        float error = fabs (botBoundsBaseline[i].x_min - botBoundsParallel[i].x_min) +
                fabs (botBoundsBaseline[i].y_min - botBoundsParallel[i].y_min) +
                fabs (botBoundsBaseline[i].z_min - botBoundsParallel[i].z_min) +
                fabs (botBoundsBaseline[i].x_max - botBoundsParallel[i].x_max) +
                fabs (botBoundsBaseline[i].y_max - botBoundsParallel[i].y_max) + 
                fabs (botBoundsBaseline[i].z_max - botBoundsParallel[i].z_max);
        
        if (error < 1E-4){
            num_correct++;
        } else {
            num_incorrect++;
            running_error+=error;
            std::cout << "Baseline x_min: " << botBoundsBaseline[i].x_min << "\tGPU x_min " << botBoundsParallel[i].x_min << std::endl; 
            std::cout << "Baseline y_min: " << botBoundsBaseline[i].y_min << "\tGPU y_min " << botBoundsParallel[i].y_min << std::endl; 
            std::cout << "Baseline z_min: " << botBoundsBaseline[i].z_min << "\tGPU z_min " << botBoundsParallel[i].z_min << std::endl; 
            std::cout << "Baseline x_max: " << botBoundsBaseline[i].x_max << "\tGPU x_max " << botBoundsParallel[i].x_max << std::endl; 
            std::cout << "Baseline y_max: " << botBoundsBaseline[i].y_max << "\tGPU y_max " << botBoundsParallel[i].y_max << std::endl; 
            std::cout << "Baseline z_max: " << botBoundsBaseline[i].z_max << "\tGPU z_max " << botBoundsParallel[i].z_max << std::endl;
        }
    }
    float avg_error = running_error / num_incorrect;
    std::cout << "Num correct AABBs: " << num_correct <<std::endl;
    std::cout << "Num incorrect AABBs: " << num_incorrect <<std::endl;
    std::cout << "Average Error " << avg_error <<std::endl;
    return true;
}

void verifyConfs(bool *confs, size_t num_confs) {
    size_t numValidConfs = 0;

    for (size_t i = 0; i < num_confs; i++) {
        if (confs[i]) numValidConfs++;
    }

    std::cout << "Valid configurations: " << numValidConfs << " (out of " << num_confs << ")" << std::endl;
}

#if(LOCAL_TESTING == 1)
void transformCPU(AABB* bot_bounds, std::vector<Configuration> &confs){
    //Load Robot
    std::vector<fcl::Vector3f> fcl_rob_vertices;
    std::vector<fcl::Triangle> fcl_rob_triangles;
    loadOBJFileFCL("./models/alpha1.0/robot.obj", fcl_rob_vertices, fcl_rob_triangles);
    std::cout << "robot has " << fcl_rob_vertices.size() << " vertices " <<std::endl;

    std::shared_ptr<fcl::BVHModel<fcl::OBBRSS<float>>> rob_mesh(new fcl::BVHModel<fcl::OBBRSS<float>>);
    rob_mesh->beginModel(fcl_rob_triangles.size(), fcl_rob_vertices.size());
    rob_mesh->addSubModel(fcl_rob_vertices, fcl_rob_triangles);
    rob_mesh->endModel();
    std::cout << "loaded robot" <<std::endl;

    fcl::Vector3f* vertices = new fcl::Vector3f[10000 * 792];

    for (int i = 0; i < confs.size(); i++){
        fcl::Transform3f transform = configurationToTransform(confs[i]);
        for (int j = 0; j < fcl_rob_vertices.size(); j++){
            vertices[i * fcl_rob_vertices.size() + j] = transform * fcl_rob_vertices[j];
        }  
    }

    generateAABBBaseline_fcl(vertices, fcl_rob_vertices.size(), confs.size(), bot_bounds);
}
#endif

//TODO: refactor code to minimize loads by interleaving file reads and device memory operations
int main()
{
      // load configurations, should have 6990 valids and 3010 invalids
    std::vector<Configuration> confs;
    readConfigurationFromFile(CONF_FILE, confs);

    Vector3f* gpu_transformed_vertices = new Vector3f[10000 * 792];
    AABB* bot_bounds_GPU = new AABB[confs.size()];

    bool *valid_conf = new bool[confs.size()];
   
    #if(LOCAL_TESTING == 1)
    AABB* bot_bounds_CPU = new AABB[confs.size()];
    fcl::Vector3f* cpu_transformed_vertices = new fcl::Vector3f[10000 * 792];

    std::chrono::time_point<std::chrono::high_resolution_clock> cpu_start_time, cpu_end_time;
    cpu_start_time = std::chrono::high_resolution_clock::now();
    transformCPU(bot_bounds_CPU, confs);
    cpu_end_time = std::chrono::high_resolution_clock::now();
    auto cpu_elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end_time - cpu_start_time);
    std::cout << "Transformation cpu execution time: " << cpu_elapsed_time.count() << " milliseconds" << std::endl;
    #endif

    std::chrono::time_point<std::chrono::high_resolution_clock> start_time, end_time;
    start_time = std::chrono::high_resolution_clock::now();

    broadPhaseFused(confs, valid_conf, bot_bounds_GPU);

    end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    std::cout << "Transformation GPU execution time: " << elapsed_time.count() << " milliseconds" << std::endl;

    
    // int num_correct = 0;
    // int num_incorrect = 0;
    // float total_error_incorrect = 0;
    // for (int i = 0; i < 10000; i++){
    //   for (int j = 0; j < 792; j++){
    //     if (verticesEqual(gpu_transformed_vertices[i * 792 + j], cpu_transformed_vertices[i * 792 + j])){
    //       num_correct++;
    //     } else {
    //       num_incorrect++;
    //     }
    //   }  
    // }

    // std::cout << "num correct is " << num_correct << std::endl;
    // std::cout << "num incorrect is " << num_incorrect << std::endl;
    // std::cout << "avg incorrect error is " << total_error_incorrect / num_incorrect << std::endl;

    verifyConfs(valid_conf, confs.size());

    #if(LOCAL_TESTING == 1)
    if(verify_generateAABB(bot_bounds_CPU, bot_bounds_GPU, confs.size()))
        std::cout << "[PASS] Parallel AABB generation matches serial generation." << std::endl;
    else
        std::cout << "[FAIL] Parallel AABB generation does not match serial generation." << std::endl;
    delete[](bot_bounds_CPU);
    #endif

    // delete[](gpu_transformed_vertices);
    delete[](bot_bounds_GPU);
    delete[](valid_conf);
}
